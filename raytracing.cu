#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <unordered_map>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <cub/device/device_radix_sort.cuh>

#include "common.cuh"
#include "math.cuh"
#include "scene.cuh"
#include "random.cuh"

__constant__ Scene cuda_scene;

__global__ void cuda_generate_initial_rays(RayData *ray_data, unsigned int *ray_indices, unsigned int *ray_keys, int rays_per_pixel, int seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    cuda_scene.generate_initial_rays(ray_data, ray_indices, ray_keys, rays_per_pixel, index, seed);
}

__global__ void cuda_process_rays(RayData *ray_data, unsigned int *ray_indices, unsigned int *keys, int ray_count, int seed)
{
    int ray_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (ray_index < ray_count)
    {
        xor_random rng;
        xor_srand(&rng, ray_index * 4137874753 + 279220567 * seed);

        cuda_scene.process_ray(ray_data + ray_indices[ray_index], keys + ray_index, rng);
    }
}

__global__ void cuda_accumulate_rays(Vec3 *framebuffer, RayData *ray_data, int rays_per_pixel)
{
    int ray_index = blockIdx.x * blockDim.x + threadIdx.x;
    int framebuffer_index = ray_index / rays_per_pixel;

    atomicAdd(&framebuffer[framebuffer_index].x, ray_data[ray_index].collected_color.x);
    atomicAdd(&framebuffer[framebuffer_index].y, ray_data[ray_index].collected_color.y);
    atomicAdd(&framebuffer[framebuffer_index].z, ray_data[ray_index].collected_color.z);
}

#define MAX_RAYS_PER_PIXEL_PER_PASS 20

void accumulate_rays_to_framebuffer(Vec3 *framebuffer, RayData *ray_data, int total_rays, int rays_per_pixel)
{
    for (int i = 0; i < total_rays; i++)
    {
        framebuffer[i / rays_per_pixel] += ray_data[i].collected_color;
    }
}

Vec3 *cpu_raytrace(Scene *scene)
{
    auto start_time = std::chrono::high_resolution_clock::now();
    int remaining_rays = scene->ray_count;

    RayData *ray_data = new RayData[scene->width * scene->height * MAX_RAYS_PER_PIXEL_PER_PASS];
    Vec3 *framebuffer = new Vec3[scene->width * scene->height]{};

    while (remaining_rays)
    {
        int rays_to_cast = min(remaining_rays, MAX_RAYS_PER_PIXEL_PER_PASS);
        remaining_rays -= rays_to_cast;

        int total_rays = rays_to_cast * scene->width * scene->height;
        #pragma omp parallel for schedule(dynamic, 1000)
        for (int i = 0; i < total_rays; i++)
        {
            scene->generate_initial_rays(ray_data, nullptr, nullptr, rays_to_cast, i, remaining_rays);
        }

        for (int i = 0; i < scene->bounces; i++)
        {
            #pragma omp parallel for schedule(dynamic, 1000)
            for (int i = 0; i < total_rays; i++)
            {
                xor_random rng;
                xor_srand(&rng, 1905678123 * i + 345903 * (remaining_rays * MAX_RAYS_PER_PIXEL_PER_PASS + i));
                scene->process_ray(&ray_data[i], nullptr, rng);
            }
        }

        accumulate_rays_to_framebuffer(framebuffer, ray_data, total_rays, rays_to_cast);
    }

    delete[] ray_data;

    auto end_time = std::chrono::high_resolution_clock::now();
    auto cpu_time = std::chrono::duration<float>(end_time - start_time).count();
    std::cout << "CPU Took " << cpu_time << "s\n";

    return framebuffer;
}

Vec3 *gpu_raytrace(const Scene *scene, bool sort)
{
    auto start_time = std::chrono::high_resolution_clock::now();

    int pixel_count = scene->width * scene->height;
    Vec3 *cuda_framebuffer;
    CUDA_CHECK(hipMalloc(&cuda_framebuffer, pixel_count   * sizeof(Vec3)));

    int max_ray_count = pixel_count * MAX_RAYS_PER_PIXEL_PER_PASS;
    RayData *cuda_ray_data;
    CUDA_CHECK(hipMalloc(&cuda_ray_data, max_ray_count * sizeof(RayData)));

    unsigned int *cuda_ray_keys[2];
    CUDA_CHECK(hipMalloc(&cuda_ray_keys[0],    max_ray_count * sizeof(unsigned int)));

    unsigned int *cuda_ray_indices[2];
    CUDA_CHECK(hipMalloc(&cuda_ray_indices[0], max_ray_count * sizeof(unsigned int)));

    size_t cuda_sort_temp_storage_size = 0;
    void *cuda_sort_temp_storage;
    if (sort)
    {
        CUDA_CHECK(hipMalloc(&cuda_ray_keys[1],    max_ray_count * sizeof(unsigned int)));
        CUDA_CHECK(hipMalloc(&cuda_ray_indices[1], max_ray_count * sizeof(unsigned int)));
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, cuda_sort_temp_storage_size, cuda_ray_keys[0], cuda_ray_keys[1], 
                                                   cuda_ray_indices[0], cuda_ray_indices[1], max_ray_count));
        CUDA_CHECK(hipMalloc(&cuda_sort_temp_storage, cuda_sort_temp_storage_size));
    }

    hipStream_t scene_copy_stream;
    hipEvent_t scene_copy_done;
    hipStream_t framebuffer_stream;
    hipEvent_t framebuffer_done;

    CUDA_CHECK(hipStreamCreate(&scene_copy_stream));
    CUDA_CHECK(hipEventCreateWithFlags(&scene_copy_done, hipEventDisableTiming));
    CUDA_CHECK(hipStreamCreate(&framebuffer_stream));
    CUDA_CHECK(hipEventCreateWithFlags(&framebuffer_done, hipEventDisableTiming));

    cuda_scene.copy_from_cpu_async(*scene, scene_copy_stream);
    hipEventRecord(scene_copy_done, scene_copy_stream);

    hipMemsetAsync(cuda_framebuffer, 0, pixel_count * sizeof(Vec3), framebuffer_stream);
    hipEventRecord(framebuffer_done, framebuffer_stream);

    int remaining_rays = scene->ray_count;

    while (remaining_rays)
    {
        int rays_to_cast = min(remaining_rays, MAX_RAYS_PER_PIXEL_PER_PASS);
        remaining_rays -= rays_to_cast;

        int total_rays = rays_to_cast * scene->width * scene->height;
        cuda_generate_initial_rays<<<(total_rays + 127) / 128, 128 >>>(cuda_ray_data, cuda_ray_indices[0], cuda_ray_keys[0], rays_to_cast, remaining_rays);

        
        for (int i = 0; i < scene->bounces; i++)
        {
            hipStreamWaitEvent(0, scene_copy_done);
            cuda_process_rays<<<(total_rays + 127) / 128, 128>>>(cuda_ray_data, cuda_ray_indices[0], cuda_ray_keys[0], total_rays, remaining_rays * MAX_RAYS_PER_PIXEL_PER_PASS + i);

            if (sort && i + 1 != scene->bounces)
            {
                CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(cuda_sort_temp_storage, cuda_sort_temp_storage_size, 
                    cuda_ray_keys[0], cuda_ray_keys[1], 
                    cuda_ray_indices[0], cuda_ray_indices[1],
                    total_rays));

                std::swap(cuda_ray_indices[0], cuda_ray_indices[1]);
                std::swap(cuda_ray_keys[0], cuda_ray_keys[1]);
            }

        }

        hipStreamWaitEvent(0, framebuffer_done);
        cuda_accumulate_rays<<<(total_rays + 127) / 128, 128>>>(cuda_framebuffer, cuda_ray_data, rays_to_cast);
    }

    Vec3 *framebuffer = new Vec3[pixel_count];

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(framebuffer, cuda_framebuffer, pixel_count * sizeof(Vec3), hipMemcpyDeviceToHost));

    cuda_scene.free_from_gpu();

    CUDA_CHECK(hipFree(cuda_framebuffer));
    CUDA_CHECK(hipFree(cuda_ray_data));
    CUDA_CHECK(hipFree(cuda_ray_keys[0]));
    CUDA_CHECK(hipFree(cuda_ray_indices[0]));
    if (sort)
    {
        CUDA_CHECK(hipFree(cuda_ray_keys[1]));
        CUDA_CHECK(hipFree(cuda_ray_indices[1]));
        CUDA_CHECK(hipFree(cuda_sort_temp_storage));
    }

    hipEventDestroy(framebuffer_done);
    hipEventDestroy(scene_copy_done);
    hipStreamDestroy(framebuffer_stream);
    hipStreamDestroy(scene_copy_stream);
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto gpu_time = std::chrono::duration<float>(end_time - start_time).count();
    std::cout << "GPU Took " << gpu_time << "s\n";

    return framebuffer;
}

void write_framebuffer_to_output_image(Scene *scene, std::vector<unsigned char> &output_image, const Vec3 *framebuffer)
{
    for (int pixel_index = 0; pixel_index < scene->width * scene->height; pixel_index++)
    {
        auto pixel = (scene->exposure / scene->ray_count) * framebuffer[pixel_index];

        float r = pixel.x;
        float g = pixel.y;
        float b = pixel.z;

        output_image.push_back((unsigned char) (sqrtf(r / (r + 1)) * 255.999f));
        output_image.push_back((unsigned char) (sqrtf(g / (g + 1)) * 255.999f));
        output_image.push_back((unsigned char) (sqrtf(b / (b + 1)) * 255.999f));
    }
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        std::cout << "Usage: " << argv[0] << " <scene>\n";
        return 1;
    }

    bool sort = true;
    bool cpu = false;
    bool gpu = true;
    bool bvh = true;

    for (int i = 2; i < argc; i++)
    {
        if (strcmp(argv[i], "no_sort") == 0)
        {
            sort = false;
        }
        else if (strcmp(argv[i], "cpu") == 0)
        {
            cpu = true;
        }
        else if (strcmp(argv[i], "no_bvh") == 0)
        {
            bvh = false;
        }
        else if (strcmp(argv[i], "no_gpu") == 0)
        {
            gpu = false;
        }
    }

    if (!cpu && !gpu)
    {
        std::cout << "No raytracing hardware specified\n";
        return 2;
    }

    Scene scene = {};
    load_scene(&scene, argv[1], bvh);
    
    std::vector<unsigned char> output_image;
    if (cpu)
    {
        Vec3 *framebuffer = cpu_raytrace(&scene);

        write_framebuffer_to_output_image(&scene, output_image, framebuffer);
        delete[] framebuffer;
    }

    if (gpu)
    {
        hipFuncAttributes attribs;
        hipFuncGetAttributes(&attribs, reinterpret_cast<const void*>(cuda_generate_initial_rays));
        hipFuncGetAttributes(&attribs, reinterpret_cast<const void*>(cuda_process_rays));
        hipFuncGetAttributes(&attribs, reinterpret_cast<const void*>(cuda_accumulate_rays));

        Vec3 *framebuffer = gpu_raytrace(&scene, sort);

        write_framebuffer_to_output_image(&scene, output_image, framebuffer);
        delete[] framebuffer;
    }

    stbi_write_png("raytracing.png", scene.width, output_image.size() / scene.width / 3, 3, &output_image.front(), scene.width * 3);

    return 0;
}