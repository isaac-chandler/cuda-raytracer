#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <unordered_map>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <cub/device/device_radix_sort.cuh>

#include "common.cuh"
#include "math.cuh"
#include "scene.cuh"
#include "random.cuh"

__constant__ Scene cuda_scene;


__device__ bool IntersectRaySphere(const Ray& ray, const Sphere& sphere, float& t)
{
    Vec3 oc = ray.origin - sphere.center;
    float a = dot(ray.direction, ray.direction);
    float b = 2.0f * dot(oc, ray.direction);
    float c = dot(oc, oc) - sphere.radius * sphere.radius;

    float discriminant = b * b - 4.0f * a * c;

    if (discriminant > 0.0f)
    {
        float t0 = (-b - sqrtf(discriminant)) / (2.0f * a);
        float t1 = (-b + sqrtf(discriminant)) / (2.0f * a);

        if (t0 > 0.0f)
        {
            t = t0;
            return true;
        }
        else if (t1 > 0.0f)
        {
            t = t1;
            return true;
        }
    }

    return false;
}


__device__ Vec3 ComputeHitPoint(const Ray& ray)
{
    // Initialize the hit point to an invalid value
    Vec3 hit_point = {FLT_MAX, FLT_MAX, FLT_MAX};

    // Iterate over all spheres in the scene
    for (int i = 0; i < cuda_scene.sphere_count; i++)
    {
        const Sphere& sphere = cuda_scene.spheres[i];
        // Perform ray-sphere intersection test
        // Implemented the intersection algorithm  above  it is a very simple algo
        float t;
        if (IntersectRaySphere(ray, sphere, t))
        {
            // Update the hit point if the new intersection point is closer
            if (t < hit_point.x)
            {
                hit_point = ray.origin + (t* ray.direction );
            }
        }
    }

    return hit_point;
}
__device__ uint32_t InterleaveBits(uint32_t value)   // 16 bit 0 to 1
{
    value = (value | (value << 16)) & 0x0000FFFF;
    value = (value | (value << 8)) & 0x00FF00FF;
    value = (value | (value << 4)) & 0x0F0F0F0F;
    value = (value | (value << 2)) & 0x33333333;
    value = (value | (value << 1)) & 0x55555555;

    return value;
}

__device__ uint64_t CalculateZCurvePosition(const Vec3& hit_point)
{
    

    // Normalize the hit point coordinates to the range [0, 1]
    float x = (hit_point.x - cuda_scene.min_coord.x) * cuda_scene.inv_dimensions.x;
    float y = (hit_point.y - cuda_scene.min_coord.y) * cuda_scene.inv_dimensions.y;
    float z = (hit_point.z - cuda_scene.min_coord.z) * cuda_scene.inv_dimensions.z;

    // Convert the normalized coordinates to integer values in the range [0, UINT_MAX]
    uint32_t xi = static_cast<uint32_t>(x * UINT_MAX);
    uint32_t yi = static_cast<uint32_t>(y * UINT_MAX);
    uint32_t zi = static_cast<uint32_t>(z * UINT_MAX);

    // Interleave the bits of the x, y, and z coordinates to create the Z-curve position
    uint64_t position = InterleaveBits(xi) | (InterleaveBits(yi) << 1) | (InterleaveBits(zi) << 2);

    return position;
}


__device__ bool CompareHitPoints(const RayData& ray1, const RayData& ray2)
{
    // Compute the Z-curve position for each ray's hit point

    // Extract the hit points from the ray data
    Vec3 hit_point1 = ray1.ray.origin + ray1.ray.direction * ray1.hit_point;  // may be ray1_pint is wrong 
    Vec3 hit_point2 = ray2.ray.origin + ray2.ray.direction * ray2.hit_point;  // same issue may be there

    // Calculate the Z-curve position based on the hit points
    uint64_t position1 = CalculateZCurvePosition(hit_point1);
    uint64_t position2 = CalculateZCurvePosition(hit_point2);

    // Compare the Z-curve positions
    return position1 < position2;
}
__device__ void ReorderRays(RayData* ray_data, unsigned int* ray_indices, unsigned int* keys, int ray_count)
{
    for (int i = 1; i < ray_count; ++i)
    {
        RayData current_ray_data = ray_data[i];
        unsigned int current_key = keys[i];
        unsigned int current_ray_index = ray_indices[i];

        int j = i - 1;
        while (j >= 0 && CompareHitPoints(ray_data[j], current_ray_data))
        {
            ray_data[j + 1] = ray_data[j];
            keys[j + 1] = keys[j];
            ray_indices[j + 1] = ray_indices[j];
            --j;
        }

        ray_data[j + 1] = current_ray_data;
        keys[j + 1] = current_key;
        ray_indices[j + 1] = current_ray_index;
    }
}




__global__ void cuda_generate_initial_rays(RayData *ray_data, unsigned int *ray_indices, unsigned int *ray_keys, int rays_per_pixel, int seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    cuda_scene.generate_initial_rays(ray_data, ray_indices, ray_keys, rays_per_pixel, index, seed);
    ray_data[index].hit_point = ComputeHitPoint(ray_data[index].ray); //ComputeHitPoint with  intersection computation 
}

__global__ void cuda_process_rays(RayData *ray_data, unsigned int *ray_indices, unsigned int *keys, int ray_count, int seed)
{
    int ray_index = blockIdx.x * blockDim.x + threadIdx.x;
    ReorderRays(ray_data, ray_indices, keys, ray_count); // `ReorderRays`
    if (ray_index < ray_count)
    {
        xor_random rng;
        xor_srand(&rng, ray_index * 4137874753 + 279220567 * seed);

        cuda_scene.process_ray(ray_data + ray_indices[ray_index], keys + ray_index, rng);
    }
    //ReorderRays(ray_data, ray_indices, keys, ray_count); // `ReorderRays` 
}

__global__ void cuda_accumulate_rays(Vec3 *framebuffer, RayData *ray_data, int rays_per_pixel, int pixel_count)
{
    int ray_index = blockIdx.x * blockDim.x + threadIdx.x;
    int framebuffer_index = ray_index / rays_per_pixel;

    if (framebuffer_index < pixel_count)
    {
        atomicAdd(&framebuffer[framebuffer_index].x, ray_data[ray_index].collected_color.x);
        atomicAdd(&framebuffer[framebuffer_index].y, ray_data[ray_index].collected_color.y);
        atomicAdd(&framebuffer[framebuffer_index].z, ray_data[ray_index].collected_color.z);
    }
}

#define MAX_RAYS_PER_PIXEL_PER_PASS 20
#define GENERATE_RAYS_BLOCK_SIZE 128
#define PROCESS_RAYS_BLOCK_SIZE 128
#define ACCUMULATE_RAYS_BLOCK_SIZE 128

void accumulate_rays_to_framebuffer(Vec3 *framebuffer, RayData *ray_data, int total_rays, int rays_per_pixel)
{
    for (int i = 0; i < total_rays; i++)
    {
        framebuffer[i / rays_per_pixel] += ray_data[i].collected_color;
    }
}

Vec3 *cpu_raytrace(Scene *scene)
{
    auto start_time = std::chrono::high_resolution_clock::now();
    int remaining_rays = scene->ray_count;

    RayData *ray_data = new RayData[scene->width * scene->height * MAX_RAYS_PER_PIXEL_PER_PASS];
    Vec3 *framebuffer = new Vec3[scene->width * scene->height]{};

    while (remaining_rays)
    {
        int rays_to_cast = min(remaining_rays, MAX_RAYS_PER_PIXEL_PER_PASS);
        remaining_rays -= rays_to_cast;

        int total_rays = rays_to_cast * scene->width * scene->height;
        #pragma omp parallel for schedule(dynamic, 1000)
        for (int i = 0; i < total_rays; i++)
        {
            scene->generate_initial_rays(ray_data, nullptr, nullptr, rays_to_cast, i, remaining_rays);
        }

        for (int i = 0; i < scene->bounces; i++)
        {
            #pragma omp parallel for schedule(dynamic, 1000)
            for (int i = 0; i < total_rays; i++)
            {
                xor_random rng;
                xor_srand(&rng, 1905678123 * i + 345903 * (remaining_rays * MAX_RAYS_PER_PIXEL_PER_PASS + i));
                scene->process_ray(&ray_data[i], nullptr, rng);
            }
        }

        accumulate_rays_to_framebuffer(framebuffer, ray_data, total_rays, rays_to_cast);
    }

    delete[] ray_data;

    auto end_time = std::chrono::high_resolution_clock::now();
    auto cpu_time = std::chrono::duration<float>(end_time - start_time).count();
    std::cout << "CPU Took " << cpu_time << "s\n";

    return framebuffer;
}

int ceil_divide(int numerator, int divisor)
{
    return (numerator + divisor - 1) / divisor;
}

Vec3 *gpu_raytrace(const Scene *scene, bool sort)
{
    auto start_time = std::chrono::high_resolution_clock::now();

    int pixel_count = scene->width * scene->height;
    Vec3 *cuda_framebuffer;
    CUDA_CHECK(hipMalloc(&cuda_framebuffer, pixel_count * sizeof(Vec3)));

    int max_ray_count = pixel_count * MAX_RAYS_PER_PIXEL_PER_PASS;
    RayData *cuda_ray_data;
    CUDA_CHECK(hipMalloc(&cuda_ray_data, max_ray_count * sizeof(RayData)));

    unsigned int *cuda_ray_keys[2];
    CUDA_CHECK(hipMalloc(&cuda_ray_keys[0], max_ray_count * sizeof(unsigned int)));

    unsigned int *cuda_ray_indices[2];
    CUDA_CHECK(hipMalloc(&cuda_ray_indices[0], max_ray_count * sizeof(unsigned int)));

    size_t cuda_sort_temp_storage_size = 0;
    void *cuda_sort_temp_storage;
    if (sort)
    {
        CUDA_CHECK(hipMalloc(&cuda_ray_keys[1],    max_ray_count * sizeof(unsigned int)));
        CUDA_CHECK(hipMalloc(&cuda_ray_indices[1], max_ray_count * sizeof(unsigned int)));
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, cuda_sort_temp_storage_size, cuda_ray_keys[0], cuda_ray_keys[1], 
                                                   cuda_ray_indices[0], cuda_ray_indices[1], max_ray_count));
        CUDA_CHECK(hipMalloc(&cuda_sort_temp_storage, cuda_sort_temp_storage_size));
    }


    hipStream_t scene_copy_stream;
    CUDA_CHECK(hipStreamCreate(&scene_copy_stream));

    hipEvent_t scene_copy_done;
    CUDA_CHECK(hipEventCreateWithFlags(&scene_copy_done, hipEventDisableTiming));

    hipStream_t framebuffer_stream;
    CUDA_CHECK(hipStreamCreate(&framebuffer_stream));

    hipEvent_t framebuffer_done;
    CUDA_CHECK(hipEventCreateWithFlags(&framebuffer_done, hipEventDisableTiming));

    // Scene copying doesn't need to finish until cuda_process_rays
    cuda_scene.copy_from_cpu_async(*scene, scene_copy_stream);
    hipEventRecord(scene_copy_done, scene_copy_stream);

    // Framebuffer doesn't need to finish zeroing until cuda_accumulate_rays
    hipMemsetAsync(cuda_framebuffer, 0, pixel_count * sizeof(Vec3), framebuffer_stream);
    hipEventRecord(framebuffer_done, framebuffer_stream);

    int remaining_rays = scene->ray_count;

    while (remaining_rays)
    {
        int rays_to_cast = min(remaining_rays, MAX_RAYS_PER_PIXEL_PER_PASS);
        remaining_rays -= rays_to_cast;

        int total_rays = rays_to_cast * scene->width * scene->height;
        cuda_generate_initial_rays<<<ceil_divide(total_rays, GENERATE_RAYS_BLOCK_SIZE), GENERATE_RAYS_BLOCK_SIZE>>>
                (cuda_ray_data, cuda_ray_indices[0], cuda_ray_keys[0], rays_to_cast, remaining_rays);

        
        for (int i = 0; i < scene->bounces; i++)
        {
            hipStreamWaitEvent(0, scene_copy_done);
            cuda_process_rays<<<ceil_divide(total_rays, PROCESS_RAYS_BLOCK_SIZE), PROCESS_RAYS_BLOCK_SIZE>>>
                    (cuda_ray_data, cuda_ray_indices[0], cuda_ray_keys[0], total_rays, remaining_rays * MAX_RAYS_PER_PIXEL_PER_PASS + i);

            if (sort && i + 1 != scene->bounces)
            {
                CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(cuda_sort_temp_storage, cuda_sort_temp_storage_size, 
                    cuda_ray_keys[0], cuda_ray_keys[1], 
                    cuda_ray_indices[0], cuda_ray_indices[1],
                    total_rays));

                std::swap(cuda_ray_indices[0], cuda_ray_indices[1]);
                std::swap(cuda_ray_keys[0], cuda_ray_keys[1]);
            }

        }

        hipStreamWaitEvent(0, framebuffer_done);
        cuda_accumulate_rays<<<ceil_divide(total_rays, ACCUMULATE_RAYS_BLOCK_SIZE), ACCUMULATE_RAYS_BLOCK_SIZE>>>
                (cuda_framebuffer, cuda_ray_data, rays_to_cast, pixel_count);
    }

    Vec3 *framebuffer = new Vec3[pixel_count];

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(framebuffer, cuda_framebuffer, pixel_count * sizeof(Vec3), hipMemcpyDeviceToHost));

    cuda_scene.free_from_gpu();

    CUDA_CHECK(hipFree(cuda_framebuffer));
    CUDA_CHECK(hipFree(cuda_ray_data));
    CUDA_CHECK(hipFree(cuda_ray_keys[0]));
    CUDA_CHECK(hipFree(cuda_ray_indices[0]));
    if (sort)
    {
        CUDA_CHECK(hipFree(cuda_ray_keys[1]));
        CUDA_CHECK(hipFree(cuda_ray_indices[1]));
        CUDA_CHECK(hipFree(cuda_sort_temp_storage));
    }

    hipEventDestroy(framebuffer_done);
    hipEventDestroy(scene_copy_done);
    hipStreamDestroy(framebuffer_stream);
    hipStreamDestroy(scene_copy_stream);
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto gpu_time = std::chrono::duration<float>(end_time - start_time).count();
    std::cout << "GPU Took " << gpu_time << "s\n";

    return framebuffer;
}

void write_framebuffer_to_output_image(Scene *scene, std::vector<unsigned char> &output_image, const Vec3 *framebuffer)
{
    for (int pixel_index = 0; pixel_index < scene->width * scene->height; pixel_index++)
    {
        auto pixel = (scene->exposure / scene->ray_count) * framebuffer[pixel_index];

        float r = pixel.x;
        float g = pixel.y;
        float b = pixel.z;

        // Convert HDR float with arbitrary range to 0-255 byte
        // x / (x + 1) does HDR to SDR tone mapping (this is a very basic way to do it)
        // Square root applies approximate linear -> sRGB conversion
        output_image.push_back((unsigned char) (sqrtf(r / (r + 1)) * 255.999f));
        output_image.push_back((unsigned char) (sqrtf(g / (g + 1)) * 255.999f));
        output_image.push_back((unsigned char) (sqrtf(b / (b + 1)) * 255.999f));
    }
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        std::cout << "Usage: " << argv[0] << " <scene>\n";
        return 1;
    }

    bool sort = true;
    bool cpu = false;
    bool gpu = true;
    bool bvh = true;

    for (int i = 2; i < argc; i++)
    {
        if (strcmp(argv[i], "no_sort") == 0)
        {
            sort = false;
        }
        else if (strcmp(argv[i], "cpu") == 0)
        {
            cpu = true;
        }
        else if (strcmp(argv[i], "no_bvh") == 0)
        {
            bvh = false;
        }
        else if (strcmp(argv[i], "no_gpu") == 0)
        {
            gpu = false;
        }
    }

    if (!cpu && !gpu)
    {
        std::cout << "No raytracing hardware specified\n";
        return 2;
    }

    Scene scene = {};
    load_scene(&scene, argv[1], bvh);
    
    std::vector<unsigned char> output_image;
    if (cpu)
    {
        Vec3 *framebuffer = cpu_raytrace(&scene);

        write_framebuffer_to_output_image(&scene, output_image, framebuffer);
        delete[] framebuffer;
    }

    if (gpu)
    {
        Vec3 *framebuffer = gpu_raytrace(&scene, sort);

        write_framebuffer_to_output_image(&scene, output_image, framebuffer);
        delete[] framebuffer;
    }

    stbi_write_png("raytracing.png", scene.width, output_image.size() / scene.width / 3, 3, &output_image.front(), scene.width * 3);

    return 0;
}