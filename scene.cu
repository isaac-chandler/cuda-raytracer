#include "hip/hip_runtime.h"
#include "scene.cuh"

#include <unordered_map>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

#define MAX_BVH_DEPTH 30

// By using this we promise CUDA that the value we are reading will never be written by a kernel
// This allows the data to be loaded into L1 cache which is not coherent
template<typename T> COMMON T load_read_only(T *t)
{
    static_assert(alignof(T) >= sizeof(float4), "load_read_only requires 16 byte alignment");

#ifdef __CUDA_ARCH__
    constexpr int count = sizeof(T) / sizeof(float4);

    union Dummy{
        float4 dummy[count];
        T value;

        COMMON Dummy() {}
    };
    
    Dummy dummy;

    #pragma unroll
    for (int i = 0; i < count; i++)
    {
        dummy.dummy[i] = __ldg(((float4 *) t) + i);
    }

    return dummy.value;
#else
    return *t;
#endif
}

// Convert number of the form 0bABCDE to
// 0b00A00B00C00D00E
__device__ unsigned short interleave_5(unsigned short x)
{
    x = (x | (x << 8)) & 0b1000000001111;
    x = (x | (x << 4)) & 0x1000010100011;
    x = (x | (x << 2)) & 0b1001001001001;

    return x;
}

__device__ unsigned short morton_code(const Vec3 &vec)
{
    unsigned short x = (unsigned short) (vec.x * 31.99);
    unsigned short y = (unsigned short) (vec.y * 31.99);
    unsigned short z = (unsigned short) (vec.z * 31.99);

    return interleave_5(x) | (interleave_5(y) << 1) | (interleave_5(z) << 2);
}

void Scene::precompute_camera_data()
{
    Vec3 right = cross(up, forward);

    float near_plane_height = 2.0f * std::tan(vertical_fov * 0.5f);
    float near_plane_width  = near_plane_height * width / height;

    scaled_right = near_plane_width * right;
    scaled_up = near_plane_height * up;

    near_plane_top_left = forward - 0.5f * scaled_right + 0.5f * scaled_up;

    inv_width = 1.0f / (width - 1);
    inv_height = 1.0f / (height - 1);
}

COMMON void Scene::generate_initial_rays(RayData *ray_data, unsigned int *ray_indices, unsigned int *ray_keys, int rays_per_pixel, int ray_index, int seed) const
{
    xor_random rng;
    xor_srand(&rng, ray_index * 298592570346 + 709579 * seed);

    int framebuffer_index = ray_index / rays_per_pixel;

    int x = framebuffer_index % width;
    int y = framebuffer_index / width;

    if (y < height)
    {
#ifdef __CUDA_ARCH__
            ray_indices[ray_index] = ray_index;
            ray_keys[ray_index] = 0;
#endif
        float x_clamped = (x + random01(&rng)) * inv_width;
        float y_clamped = (y + random01(&rng)) * inv_height;

        RayData ray;

        ray.ray = {camera_position, normalise(near_plane_top_left + x_clamped * scaled_right - y_clamped * scaled_up)};
        ray.transmitted_color = {1, 1, 1};
        ray.collected_color = {0, 0, 0};

        ray_data[ray_index] = ray;
    }
}

// Branchless ray AABB intersection from https://tavianator.com/2022/ray_box_boundary.html
// (assuming there is hardware min/max which is true on all modern GPUs and CPUs)
COMMON bool ray_aabb_intersection(const Aabb &aabb, const Ray &ray, const Vec3 &n_inv, float &tmin, float tmax)
{
    tmin = 0.0f;

    float t1 = (aabb.min_bound.x - ray.origin.x) * n_inv.x;
    float t2 = (aabb.max_bound.x - ray.origin.x) * n_inv.x;

    tmin = min(max(t1, tmin), max(t2, tmin));
    tmax = max(min(t1, tmax), min(t2, tmax));

    t1 = (aabb.min_bound.y - ray.origin.y) * n_inv.y;
    t2 = (aabb.max_bound.y - ray.origin.y) * n_inv.y;

    tmin = min(max(t1, tmin), max(t2, tmin));
    tmax = max(min(t1, tmax), min(t2, tmax));

    t1 = (aabb.min_bound.z - ray.origin.z) * n_inv.z;
    t2 = (aabb.max_bound.z - ray.origin.z) * n_inv.z;

    tmin = min(max(t1, tmin), max(t2, tmin));
    tmax = max(min(t1, tmax), min(t2, tmax));

    return tmin <= tmax;
}

COMMON void Scene::bvh_closest_hit_distance(const Ray &ray, float &closest_hit_distance, int &closest_hit_index) const
{
    Vec3 n_inv = {1 / ray.direction.x, 1 / ray.direction.y, 1 / ray.direction.z};

    unsigned int node_index_stack[MAX_BVH_DEPTH + 1];
    float node_distance_stack[MAX_BVH_DEPTH + 1];
    int stack_count = 1;

    node_index_stack[0] = 0;
    node_distance_stack[0] = 0;

    while (stack_count)
    {
        stack_count--;
        float distance = node_distance_stack[stack_count];

        if (distance >= closest_hit_distance)
        {
            continue;
        }

        BvhNode node = load_read_only(&bvh[node_index_stack[stack_count]]);

        
        if (node.is_leaf())
        {
            for (int i = node.child2; i < node.child1; i++)
            {
                const auto triangle = load_read_only(&triangles[i]);

                // Möller–Trumbore ray-triangle intersection algorithm
                // Based on https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
                Vec3 h = cross(ray.direction, triangle.p3p1);
                float perpendicular_component = dot(h, triangle.p2p1);

                float x = dot(triangle.normal, ray.direction);

                if (perpendicular_component == 0)
                    continue;

                float inv_perpendicular_component = 1 / perpendicular_component;

                Vec3 offset = ray.origin - triangle.p1;
                float u = dot(offset, h) * inv_perpendicular_component;

                if (u < 0 || u > 1)
                    continue;

                Vec3 q = cross(offset, triangle.p2p1);
                float v = dot(ray.direction, q) * inv_perpendicular_component;

                if (v < 0 || u + v > 1)
                    continue;

                float hit_distance = dot(triangle.p3p1, q) * inv_perpendicular_component;

                if (hit_distance < 0.005 || hit_distance >= closest_hit_distance)
                    continue;

                closest_hit_distance = hit_distance;
                closest_hit_index = sphere_count + i;
            }
        }
        else
        {
            float hit1_distance, hit2_distance;

            bool hit1 = ray_aabb_intersection(load_read_only(&bvh[node.child1]).aabb, ray, n_inv, hit1_distance, closest_hit_distance);
            bool hit2 = ray_aabb_intersection(load_read_only(&bvh[node.child2]).aabb, ray, n_inv, hit2_distance, closest_hit_distance);

            if (hit1 && hit2)
            {
                if (hit1_distance < hit2_distance)
                {
                    node_index_stack[stack_count] = node.child1;
                    node_distance_stack[stack_count] = hit1_distance;
                    stack_count++;

                    node_index_stack[stack_count] = node.child2;
                    node_distance_stack[stack_count] = hit2_distance;
                    stack_count++;
                }
                else
                {
                    node_index_stack[stack_count] = node.child2;
                    node_distance_stack[stack_count] = hit2_distance;
                    stack_count++;

                    node_index_stack[stack_count] = node.child1;
                    node_distance_stack[stack_count] = hit1_distance;
                    stack_count++;
                }
            }
            else if (hit1)
            {
                node_index_stack[stack_count] = node.child1;
                node_distance_stack[stack_count] = hit1_distance;
                stack_count++;
            }
            else if (hit2)
            {
                node_index_stack[stack_count] = node.child2;
                node_distance_stack[stack_count] = hit2_distance;
                stack_count++;
            }
        }
    }
}

void Scene::copy_from_cpu_async(const Scene &scene, hipStream_t stream)
{
    Scene scene_copy = scene;

    int primitive_count = scene.triangle_count + scene.sphere_count;
    int environment_map_size = scene.environment_map_width * scene.environment_map_height;

    CUDA_CHECK(hipMalloc(&scene_copy.spheres,          scene.sphere_count   * sizeof(Sphere)));
    CUDA_CHECK(hipMalloc(&scene_copy.triangles,        scene.triangle_count * sizeof(Triangle)));
    CUDA_CHECK(hipMalloc(&scene_copy.materials,        scene.material_count * sizeof(Material)));
    CUDA_CHECK(hipMalloc(&scene_copy.material_indices, primitive_count      * sizeof(uint16_t)));
    CUDA_CHECK(hipMalloc(&scene_copy.bvh,              scene.bvh_node_count * sizeof(BvhNode)));
    CUDA_CHECK(hipMalloc(&scene_copy.environment_map,  environment_map_size * sizeof(Vec3)));

    CUDA_CHECK(hipMemcpyAsync(scene_copy.spheres,          scene.spheres,          sizeof(Sphere)   * scene.sphere_count,   hipMemcpyHostToDevice, stream));    
    CUDA_CHECK(hipMemcpyAsync(scene_copy.triangles,        scene.triangles,        sizeof(Triangle) * scene.triangle_count, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(scene_copy.materials,        scene.materials,        sizeof(Material) * scene.material_count, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(scene_copy.material_indices, scene.material_indices, sizeof(uint16_t) * primitive_count,      hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(scene_copy.bvh,              scene.bvh,              sizeof(BvhNode)  * scene.bvh_node_count, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(scene_copy.environment_map,  scene.environment_map,  sizeof(Vec3)     * environment_map_size, hipMemcpyHostToDevice, stream));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(*this), &scene_copy, sizeof(Scene)));
}

void Scene::free_from_gpu()
{
    Scene scene_copy;
    CUDA_CHECK(hipMemcpyFromSymbol(&scene_copy, HIP_SYMBOL(*this), sizeof(Scene)));
    
    CUDA_CHECK(hipFree(scene_copy.environment_map));
    CUDA_CHECK(hipFree(scene_copy.material_indices));
    CUDA_CHECK(hipFree(scene_copy.materials));
    CUDA_CHECK(hipFree(scene_copy.bvh));
    CUDA_CHECK(hipFree(scene_copy.triangles));
    CUDA_CHECK(hipFree(scene_copy.spheres));
}


// Maybe it makes sense to pre-convert to a cubemap instead of doing this every time a ray misses
// Our test environment map is in a format used by the PBRTv4 ray tracer
// This code is based on https://github.com/mmp/pbrt-v4/blob/c4baa534042e2ec4eb245924efbcef477e096389/src/pbrt/util/math.cpp#L317
COMMON Vec3 equal_area_project_sphere_to_square(const Vec3 &direction)
{
    float x = abs(direction.x);
    float y = abs(direction.y);
    float z = abs(direction.z);

    float r = sqrt(1 - min(z, 1.0f));

    float a = max(x, y);
    float b = min(x, y);

    b = a == 0 ? 0 : b / a;

    float phi = (2 / M_PI) * atan(b);

    if (x < y)
    {
        phi = 1 - phi;
    }

    float v = phi * r;
    float u = r - v;

    if (direction.z < 0)
    {
        float old_v = v;
        v = 1 - u;
        u = 1 - old_v;
    }

    u = copysign(u, direction.x);
    v = copysign(v, direction.y);

    return {(u + 1) * 0.5f, (v + 1) * 0.5f, 0};
}

COMMON void Scene::process_ray(RayData *ray_data_ptr, unsigned int *ray_key, xor_random rng) const
{
#ifdef __CUDA_ARCH__
    if (*ray_key == 0xFFFF'FFFF)
        return;
#else
    if (ray_data_ptr->transmitted_color.x == 0 && ray_data_ptr->transmitted_color.y == 0 && ray_data_ptr->transmitted_color.z == 0)
        return;
#endif

    RayData ray_data = *ray_data_ptr;

    float closest_hit_distance = 1e30;

    int closest_hit_index = -1;

    const auto ray = ray_data.ray;

    for (int i = 0; i < sphere_count; i++)
    {
        const auto sphere = spheres[i];

        Vec3 offset = sphere.center - ray.origin;
        

        float minus_half_b = dot(offset, ray.direction);
        float quarter_c = magnitude_squared(offset) - sphere.radius * sphere.radius;

        float quarter_discriminant = minus_half_b * minus_half_b - quarter_c;

        if (quarter_discriminant < 0)
            continue;

        float half_square_root = sqrtf(quarter_discriminant);

        float hit_distance = minus_half_b - half_square_root;

        if (hit_distance < closest_hit_distance && hit_distance >= 0.005)
        {
            closest_hit_distance = hit_distance;
            closest_hit_index = i;
            continue;
        }

        hit_distance = minus_half_b + half_square_root;

        if (hit_distance < closest_hit_distance && hit_distance >= 0.005)
        {
            closest_hit_distance = hit_distance;
            closest_hit_index = i;
            continue;
        }
    }

    bvh_closest_hit_distance(ray, closest_hit_distance, closest_hit_index);

    if (closest_hit_index == -1)
    {
        // Environment map in our test data is rotated and has y and z axes flipped, 
        // apply a hardcoded transformation for now.
        float dir_x = ray.direction.x * -0.386527 + ray.direction.z * 0.922278;
        float dir_y = ray.direction.x * -0.922278 + ray.direction.z * -0.386527;
        float dir_z = ray.direction.y;

        Vec3 coords = equal_area_project_sphere_to_square({dir_x, dir_y, dir_z});
        float x = coords.x;
        float y = coords.y;

        // Nearest filtering
        int texel_x = (int) (clamp01(x) * (environment_map_width  - 1) + 0.5);
        int texel_y = (int) (clamp01(y) * (environment_map_height - 1) + 0.5);
        Vec3 sky_color = environment_map[texel_y * environment_map_height + texel_x];

        ray_data.collected_color += sky_color * ray_data.transmitted_color;
        ray_data.transmitted_color = {0, 0, 0};
    }
    else
    {
        const auto hit_point = ray.origin + closest_hit_distance * ray.direction;
        ray_data.ray.origin = hit_point;

        Vec3 normal;
        if (closest_hit_index < sphere_count)
        {
            const auto hit_sphere = spheres[closest_hit_index];
            normal = (1 / hit_sphere.radius) * (hit_point - hit_sphere.center);
        }
        else
        {
            const auto hit_triangle = triangles[closest_hit_index - sphere_count];
            normal = hit_triangle.normal;
        }

        const auto material = load_read_only(&materials[material_indices[closest_hit_index]]);

        ray_data.collected_color += material.emitted * ray_data.transmitted_color;
        

        bool front_face = dot(normal, ray.direction) < 0;

        if (!front_face)
        {
            normal = -normal;
        }

        Vec3 rough_normal = normalise(normal + material.roughness * random_on_sphere(&rng));
        float cos_theta = dot(rough_normal, ray.direction);



        if (material.index_of_refraction == 0)
        {
            if (random01(&rng) <= material.metallicity)
            {
                ray_data.transmitted_color *= material.specular_albedo;
                ray_data.ray.direction = ray.direction - 2 * cos_theta * rough_normal;
            }
            else
            {
                ray_data.transmitted_color *= material.diffuse_albedo;
                ray_data.ray.direction = normalise(normal + random_on_sphere(&rng)); 
            }
        }
        else
        {
            float ior = material.index_of_refraction;
            float inv_ior = 1 / ior;

            if (front_face)
            {
                float temp = inv_ior;
                inv_ior = ior;
                ior = temp;
            }

            float sin_theta_squared = 1 - cos_theta * cos_theta;

            float r0 = (1 - ior) / (1 + ior);
            r0 *= r0;

            float cosine = 1 + cos_theta;
            float reflectance = r0 + (1 - r0) * cosine * cosine * cosine * cosine * cosine;

            if (sin_theta_squared > inv_ior * inv_ior || random01(&rng) < reflectance)
            {
                ray_data.transmitted_color *= material.specular_albedo;
                ray_data.ray.direction = ray.direction - 2 * cos_theta * rough_normal;                
            }
            else
            {
                ray_data.transmitted_color *= material.diffuse_albedo;
                
                Vec3 r_out_perp = ior * (ray.direction - cos_theta * rough_normal);
                Vec3 r_out_parallel = -sqrtf(1 - magnitude_squared(r_out_perp)) * rough_normal;
                ray_data.ray.direction = normalise(r_out_parallel + r_out_perp);
            }
        }
    }


#ifdef __CUDA_ARCH__
    if (ray_data.transmitted_color.x == 0 && ray_data.transmitted_color.y == 0 && ray_data.transmitted_color.z == 0)
        *ray_key = 0xFFFF'FFFF;
    else
        *ray_key = ((unsigned int) morton_code((ray_data.ray.origin - min_coord) * inv_dimensions) << 16) | (unsigned int) morton_code(0.5 * (ray_data.ray.direction + Vec3{1, 1, 1}));
#endif
    *ray_data_ptr = ray_data;
}

// Extremely hacky ply file loader for exactly the ply format we have
// will not work for most files
void load_ply(std::vector<Triangle> &triangles, const std::string &filename)
{
    std::ifstream ply_file(filename, std::ios_base::binary);

    std::string line;
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);

    auto vertex_count = std::stoi(line.substr(15));

    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);
    std::getline(ply_file, line);

    auto face_count = std::stoi(line.substr(13));

    std::getline(ply_file, line);
    std::getline(ply_file, line);

    struct Vertex {
        Vec3 position;
        Vec3 normal;
        float u, v;
    };

    std::vector<Vertex> vertices;
    vertices.resize(vertex_count);
    ply_file.read(reinterpret_cast<char *>(&vertices.front()), sizeof(Vertex) * vertices.size());

    std::vector<int> indices;

    for (int i = 0; i < face_count; i++)
    {
        indices.resize(ply_file.get());
        ply_file.read(reinterpret_cast<char *>(&indices.front()), sizeof(int) * indices.size());

        for (int j = 2; j < indices.size(); j++)
        {
            Triangle triangle;

            triangle.p1 = vertices[indices[0]].position;
            triangle.p2p1 = vertices[indices[j - 1]].position;
            triangle.p3p1 = vertices[indices[j]].position;
            triangle.normal = (1.0f / 3.0f) * (triangle.p1 + triangle.p2p1 + triangle.p3p1);

            triangles.push_back(triangle);
        }
    }
}

Vec3 *load_pfm(const std::string &filename, int *width, int *height)
{
    std::ifstream file(filename, std::ios_base::binary);

    std::string line;
    std::getline(file, line);
    std::getline(file, line);

    std::stringstream ss(line);
    
    ss >> *width;
    ss >> *height;

    std::getline(file, line);

    Vec3 *image = new Vec3[*width * *height];
    file.read((char *) image, sizeof(Vec3) * *width * *height);

    return image;
}

void load_scene(Scene *scene, const char *filename, bool use_bvh)
{
    scene->width = 1920;
    scene->height = 1080;
    scene->ray_count = 1;
    scene->bounces = 3;

    std::vector<Sphere> spheres;
    std::vector<Triangle> triangles;

    std::ifstream scene_file(filename);

    std::unordered_map<std::string, uint16_t> materials_map;
    std::vector<Material> materials;
    std::vector<uint16_t> sphere_materials;
    std::vector<uint16_t> triangle_materials;

    for (std::string line; std::getline(scene_file, line);)
    {
        if (line.empty())
            continue;

        std::istringstream tokens(line);

        std::string token;
        std::getline(tokens, token, ' ' );


        if (token == "sky")
        {
            float r, g, b;

            tokens >> r;
            tokens >> g;
            tokens >> b;

            scene->environment_map = new Vec3{r, g, b};
            scene->environment_map_width = 1;
            scene->environment_map_height = 1;
        }
        else if (token == "sky_map")
        {
            std::getline(tokens, token, ' ' );

            scene->environment_map = load_pfm(token, &scene->environment_map_width, &scene->environment_map_height);
            std::cout << "Loaded environment map with size " << scene->environment_map_width << "," << scene->environment_map_height << "\n";

        }
        else if (token == "camera")
        {
            std::getline(tokens, token, ' ' );

            tokens >> scene->camera_position.x;
            tokens >> scene->camera_position.y;
            tokens >> scene->camera_position.z;

            std::getline(tokens, token, ' ' );
            std::getline(tokens, token, ' ' );

            tokens >> scene->forward.x;
            tokens >> scene->forward.y;
            tokens >> scene->forward.z;
            scene->forward = normalise(scene->forward);

            std::getline(tokens, token, ' ' );
            std::getline(tokens, token, ' ' );

            tokens >> scene->up.x;
            tokens >> scene->up.y;
            tokens >> scene->up.z;
            scene->up = normalise(scene->up);

            std::getline(tokens, token, ' ' );
            std::getline(tokens, token, ' ' );

            tokens >> scene->vertical_fov;
            scene->vertical_fov = scene->vertical_fov * (M_PI / 180);
        }
        else if (token == "material")
        {
            std::getline(tokens, token, ' ' );

            materials_map[token] = (uint16_t) materials.size();

            Material material;
            material.specular_albedo = {1, 1, 1};
            material.diffuse_albedo = {1, 1, 1};
            material.emitted = {0, 0, 0};
            material.metallicity = 0;
            material.roughness = 0;
            material.index_of_refraction = 0;

            while (std::getline(tokens, token, ' ' ))
            {
                if (token == "diffuse")
                {
                    tokens >> material.diffuse_albedo.x;
                    tokens >> material.diffuse_albedo.y;
                    tokens >> material.diffuse_albedo.z;
                }
                else if (token == "specular")
                {
                    tokens >> material.specular_albedo.x;
                    tokens >> material.specular_albedo.y;
                    tokens >> material.specular_albedo.z;
                }
                else if (token == "emit")
                {
                    tokens >> material.emitted.x;
                    tokens >> material.emitted.y;
                    tokens >> material.emitted.z;
                }
                else if (token == "metallicity")
                {
                    tokens >> material.metallicity;
                }
                else if (token == "roughness")
                {
                    tokens >> material.roughness;
                }
                else if (token == "ior")
                {
                    tokens >> material.index_of_refraction;
                }
            }

            materials.push_back(material);
        }
        else if (token == "sphere")
        {
            std::getline(tokens, token, ' ' );

            sphere_materials.push_back(materials_map.at(token));

            Sphere sphere;

            tokens >> sphere.center.x;
            tokens >> sphere.center.y;
            tokens >> sphere.center.z;
            tokens >> sphere.radius;

            spheres.push_back(sphere);
        }
        else if (token == "triangle")
        {
            std::getline(tokens, token, ' ' );

            triangle_materials.push_back(materials_map.at(token));

            Triangle triangle;

            tokens >> triangle.p1.x;
            tokens >> triangle.p1.y;
            tokens >> triangle.p1.z;

            tokens >> triangle.p2p1.x;
            tokens >> triangle.p2p1.y;
            tokens >> triangle.p2p1.z;

            tokens >> triangle.p3p1.x;
            tokens >> triangle.p3p1.y;
            tokens >> triangle.p3p1.z;

            triangle.normal = (1.0f / 3.0f) * (triangle.p1 + triangle.p2p1 + triangle.p3p1);

            triangles.push_back(triangle);
        }
        else if (token == "quad")
        {
            std::getline(tokens, token, ' ' );

            triangle_materials.push_back(materials_map.at(token));
            triangle_materials.push_back(materials_map.at(token));

            Vec3 p1, p2, p3, p4;

            tokens >> p1.x;
            tokens >> p1.y;
            tokens >> p1.z;

            tokens >> p2.x;
            tokens >> p2.y;
            tokens >> p2.z;

            tokens >> p3.x;
            tokens >> p3.y;
            tokens >> p3.z;

            tokens >> p4.x;
            tokens >> p4.y;
            tokens >> p4.z;

            Triangle triangle;

            triangle.p1 = p1;
            triangle.p2p1 = p2;
            triangle.p3p1 = p3;
            triangle.normal = (1.0f / 3.0f) * (triangle.p1 + triangle.p2p1 + triangle.p3p1);

            triangles.push_back(triangle);

            triangle.p1 = p1;
            triangle.p2p1 = p3;
            triangle.p3p1 = p4;
            triangle.normal = (1.0f / 3.0f) * (triangle.p1 + triangle.p2p1 + triangle.p3p1);

            triangles.push_back(triangle);
        }
        else if (token == "ply")
        {
            std::getline(tokens, token, ' ' );

            const auto material = materials_map.at(token);

            size_t triangle_count = triangles.size();

            std::getline(tokens, token, ' ' );
            load_ply(triangles, token);

            for (; triangle_count < triangles.size(); triangle_count++)
            {
                triangle_materials.push_back(material);
            }
        }
        else if (token == "image")
        {
            tokens >> scene->width;
            tokens >> scene->height;
            tokens >> scene->ray_count;
            tokens >> scene->bounces;
            tokens >> scene->exposure;
        }
    }

    scene->sphere_count = spheres.size();
    scene->spheres = new Sphere[spheres.size()];
    std::copy(spheres.begin(), spheres.end(), scene->spheres);
    
    scene->triangle_count = triangles.size();
    scene->triangles = new Triangle[triangles.size()];
    std::copy(triangles.begin(), triangles.end(), scene->triangles);

    scene->material_indices = new uint16_t[sphere_materials.size() + triangle_materials.size()];
    std::copy(sphere_materials.begin(), sphere_materials.end(), scene->material_indices);
    std::copy(triangle_materials.begin(), triangle_materials.end(), scene->material_indices + sphere_materials.size());

    scene->materials = new Material[materials.size()];
    std::copy(materials.begin(), materials.end(), scene->materials);
    scene->material_count = (uint16_t) materials.size();

    scene->precompute_camera_data();
    scene->generate_bvh(use_bvh ? MAX_BVH_DEPTH : 0);

    scene->min_coord = scene->bvh[0].aabb.min_bound;
    Vec3 scene_max_coord = scene->bvh[0].aabb.max_bound;
    for (const auto &sphere : spheres)
    {
        scene_max_coord = max(scene_max_coord, sphere.center + Vec3{sphere.radius, sphere.radius, sphere.radius});
        scene->min_coord = min(scene->min_coord, sphere.center - Vec3{sphere.radius, sphere.radius, sphere.radius});
    }

    scene->inv_dimensions = {1 / scene_max_coord.x, 1 / scene_max_coord.y, 1 / scene_max_coord.z};
}

void Aabb::expand(const Vec3 &other)
{
    min_bound = min(min_bound, other);
    max_bound = max(max_bound, other);
}

void Aabb::expand(const Triangle &other)
{
    expand(other.p1);
    expand(other.p2p1);
    expand(other.p3p1);
}

void Aabb::expand(const Aabb &other)
{
    min_bound = min(min_bound, other.min_bound);
    max_bound = max(max_bound, other.max_bound);
}

float Aabb::half_area() const
{
    Vec3 size = max_bound - min_bound;

    return size.x * size.y + size.x * size.z + size.y * size.z;
}

COMMON bool BvhNode::is_leaf() const
{
    return child2 <= child1;
}

// Binned surface area heuristic BVH computation
// Based on https://jacco.ompf2.com/2022/04/21/how-to-build-a-bvh-part-3-quick-builds/
void BvhNode::maybe_split(const Scene *scene, std::vector<BvhNode> &bvh_nodes, int max_depth)
{
    for (int i = child2; i < child1; i++)
    {
        aabb.expand(scene->triangles[i]);
    }

    int our_count = child1 - child2;

    if (our_count <= 4 || max_depth == 0)
    {
        return;
    }

    float our_cost = aabb.half_area() * our_count;

    struct Bin
    {
        Aabb aabb;
        int triangle_count = 0;
    };

    constexpr int BINS = 8;

    int best_axis;
    float best_position;
    float best_cost = our_cost;

    for (int axis = 0; axis < 3; axis++)
    {
        float min_centroid = 1e30;
        float max_centroid = -1e30;

        for (int i = child2; i < child1; i++)
        {
            const auto &triangle = scene->triangles[i];
            min_centroid = min(min_centroid, triangle.normal[axis]);
            max_centroid = max(max_centroid, triangle.normal[axis]);
        }

        if (min_centroid == max_centroid)
            continue;

        float scale = BINS / (max_centroid - min_centroid);

        Bin bins[BINS];

        for (int i = child2; i < child1; i++)
        {
            const auto &triangle = scene->triangles[i];
            auto &bin =  bins[std::min(BINS - 1, (int) ((triangle.normal[axis] - min_centroid) * scale))];

            bin.triangle_count++;

            bin.aabb.expand(triangle);
        }

        float left_area[BINS - 1], right_area[BINS - 1];
        int left_count[BINS - 1];
        int left_sum = 0;

        Aabb left_box, right_box;

        for (int i = 0; i + 1 < BINS; i++)
        {
            left_sum += bins[i].triangle_count;
            left_count[i] = left_sum;
            left_box.expand(bins[i].aabb);
            left_area[i] = left_box.half_area();

            right_box.expand(bins[BINS - 1 - i].aabb);
            right_area[BINS - 2 - i] = right_box.half_area();
        }

        scale = (max_centroid - min_centroid) / BINS;

        for (int i = 0; i + 1 < BINS; i++)
        {
            float plane_cost = left_count[i] * left_area[i] + (our_count - left_count[i]) * right_area[i];

            if (plane_cost != 0 && plane_cost < best_cost)
            {
                best_axis = axis;
                best_position = min_centroid + scale * (i + 1);
                best_cost = plane_cost;
            }
        }
    }

    if (best_cost >= our_cost)
    {
        return;
    }

    int i = child2;
    int j = child1 - 1;

    while (i <= j)
    {
        if (scene->triangles[i].normal[best_axis] < best_position)
        {
            i++;
        }
        else
        {
            std::swap(scene->triangles[i], scene->triangles[j]);
            std::swap(scene->material_indices[scene->sphere_count + i], scene->material_indices[scene->sphere_count + j]);
            j--;
        }
    }

    if (i == child1 || i == child2)
    {
        return;
    }

    int left_child_index = bvh_nodes.size();
    bvh_nodes.emplace_back();
    auto &left_child = bvh_nodes.back();

    int right_child_index = bvh_nodes.size();
    bvh_nodes.emplace_back();
    auto &right_child = bvh_nodes.back();

    left_child.child2 = child2;
    left_child.child1 = i;
    right_child.child2 = i;
    right_child.child1 = child1;

    left_child.maybe_split(scene, bvh_nodes, max_depth - 1);
    right_child.maybe_split(scene, bvh_nodes, max_depth - 1);

    child1 = left_child_index;
    child2 = right_child_index;
}

void Scene::generate_bvh(int max_depth)
{
    auto start_time = std::chrono::high_resolution_clock::now();
    decltype(start_time) end_time;

    std::vector<BvhNode> bvh_nodes;
    // Not only for speed, needed so that references will never be invalidated
    bvh_nodes.reserve(triangle_count * 2);

    bvh_nodes.emplace_back();
    auto &root = bvh_nodes.back();

    root.child2 = 0;
    root.child1 = triangle_count;
    
    root.maybe_split(this, bvh_nodes, max_depth);

    bvh_node_count = bvh_nodes.size();
    bvh = new BvhNode[bvh_nodes.size()];
    std::copy(bvh_nodes.begin(), bvh_nodes.end(), bvh);

    end_time = std::chrono::high_resolution_clock::now();
    auto bvh_time = std::chrono::duration<float>(end_time - start_time).count();
    std::cout << "Triangle count: " << triangle_count << "\n";
    std::cout << "BVH Took " << (bvh_time * 1000) << "ms\n";
    std::cout << "Node count: " << bvh_node_count << "\n";

    for (int i = 0; i < triangle_count; i++)
    {
        auto &triangle = triangles[i];
        triangle.p2p1 -= triangle.p1;
        triangle.p3p1 -= triangle.p1;
        triangle.normal = normalise(cross(triangle.p3p1, triangle.p2p1));
    }
}